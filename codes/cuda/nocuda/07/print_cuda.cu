
#include <hip/hip_runtime.h>
#include <cstdio>

#ifdef ENABLE_CUDA
__global__ void kernelPrint()
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx == 0 )
    {
        std::printf("blockDim.x.y.z=[%d,%d,%d]\n", blockDim.x, blockDim.y, blockDim.z);
        std::printf("gridDim.x.y.z=[%d,%d,%d]\n", gridDim.x, gridDim.y, gridDim.z);
    }
}

void PrintGuda()
{
    std::printf("start void PrintGuda()------------------------\n");

    kernelPrint<<<1, 10 >>>();
    kernelPrint<<<5, 10 >>>();

    cudaDeviceSynchronize();
    std::printf("end void PrintGuda()------------------------\n");
}

#endif